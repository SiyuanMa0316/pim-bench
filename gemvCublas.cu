//Example 2. Application Using C and cuBLAS: 0-based indexing
//-----------------------------------------------------------
#include <iostream>
#include <cstdlib>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define M 6
#define N 5
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

static __inline__ void gemvCublas (hipblasHandle_t handle, float*A, float*x, float* y, int m, int n){
    hipblasStatus_t stat;
    float alf = 1.0;
    float bet = 1.0;
    const float *alpha = &alf;
    const float *beta = &bet;

    stat = hipblasSgemv( handle, HIPBLAS_OP_N,
                           6, 5,
                           alpha,
                           A, 6,
                           x, 1,
                           beta,
                           y, 1);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("hipblasSgemv failed\n");
    }
    if (stat == HIPBLAS_STATUS_NOT_INITIALIZED){
        printf ("library not initialized\n");
    }
    if (stat == HIPBLAS_STATUS_INVALID_VALUE){
        printf ("the parameters m,n<0 or incx,incy=0\n");
    }
    if (stat == HIPBLAS_STATUS_EXECUTION_FAILED){
        printf ("the function failed to launch on the GPU\n");
    }
}

int main (void){
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;

    int i, j;

    float* devMat;
    float* mat = 0;
    mat = (float *)malloc (M * N * sizeof (*mat));
    if (!mat) {
        printf ("host memory allocation failed");
        return EXIT_FAILURE;
    }
    for (j = 0; j < N; j++) {
        for (i = 0; i < M; i++) {
            mat[IDX2C(i,j,M)] = (float)(i * N + j + 1);
        }
    }

    float* devX;
    float* x=0;
    x = (float *)malloc(N * sizeof(*x));
    if(!x){
        printf ("host memory allocation failed");
        return EXIT_FAILURE;
    }
    for(i=0; i<N; i++){
        x[i]=(float)(i+1);
    }

    float* devY;
    float* y=0;
    y = (float *)malloc(N * sizeof(*y));


    
    cudaStat = hipMalloc ((void**)&devMat, M*N*sizeof(*mat));
    if (cudaStat != hipSuccess) {
        printf ("matrix device memory allocation failed");
        return EXIT_FAILURE;
    }
    cudaStat = hipMalloc ((void**)&devX, N*sizeof(*x));
    if (cudaStat != hipSuccess) {
        printf ("x device memory allocation failed");
        return EXIT_FAILURE;
    }
    cudaStat = hipMalloc ((void**)&devY, N*sizeof(*y));
    if (cudaStat != hipSuccess) {
        printf ("y device memory allocation failed");
        return EXIT_FAILURE;
    }

    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }
    stat = hipblasSetMatrix (M, N, sizeof(*mat), mat, M, devMat, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("matrix data download failed");
        hipFree (devMat);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    stat = hipblasSetVector(N, sizeof(*x), x, 1, devX, 1);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("vector data download failed");
        hipFree (devX);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    stat = hipblasSetVector(N, sizeof(*x), x, 1, devY, 1);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("vector data download failed");
        hipFree (devY);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    //verify copy success
    stat = hipblasGetMatrix (M, N, sizeof(*mat), devMat, M, mat, M);
    for (j = 0; j < N; j++) {
        for (i = 0; i < M; i++) {
            printf ("%7.0f", mat[IDX2C(i,j,M)]);
        }
        printf ("\n");
    }
    stat = hipblasGetVector (N, sizeof(*x), devX, 1, x, 1);
    for (i = 0; i < N; i++) {
        printf ("%7.0f", x[i]);
    }
    printf ("\n");
    stat = hipblasGetVector (N, sizeof(*x), devY, 1, y, 1);
    for (i = 0; i < N; i++) {
        printf ("%7.0f", y[i]);
    }
    printf ("\n");

    //do the gemv
    gemvCublas(handle, devMat, devX, devY, M, N);

    //get y
    stat = hipblasGetVector (N, sizeof(*y), devY, 1, y, 1);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data upload failed");
        hipFree (devY);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    hipFree (devY);
    hipFree (devX);
    hipFree (devMat);
    hipblasDestroy(handle);

    //print y
    for (i = 0; i < N; i++) {
        printf ("%7.0f", y[i]);
    }
    printf ("\n");
    
    free(y);
    free(x);
    free(mat);
    return EXIT_SUCCESS;
}
