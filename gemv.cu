#include "hip/hip_runtime.h"
/*****************************************************************************

		C-DAC Tech Workshop : hyPACK-2013
                           October 15-18, 2013

  Example     :  cuda-matrix-vector-multiplication.cu 
 
  Objective   : Write CUDA program to compute Matrix-Vector multiplication.

  Input       : None 

  Output      : Execution time in seconds , Gflops achieved
                                                                                                                            
  Created     : August-2013

  E-mail      : hpcfte@cdac.in     
                                 
****************************************************************************/


#include<stdio.h>
#include<hip/hip_runtime.h>

#define BLOCKSIZE 256
#define SIZE 32
#define EPS 1.0e-15

hipDeviceProp_t deviceProp;	


double *host_Mat,*host_Vect,*host_ResVect,*cpu_ResVect;
double *device_Mat,*device_Vect,*device_ResVect;
int     vlength ,matRowSize , matColSize;
int     device_Count;
int     size = SIZE;

/*mem error*/
void mem_error(char *arrayname, char *benchmark, int len, char *type)
{
        printf("\nMemory not sufficient to allocate for array %s\n\tBenchmark : %s  \n\tMemory requested = %d number of %s elements\n",arrayname, benchmark, len, type);
        exit(-1);
}

/*calculate Gflops*/
double calculate_gflops(float &Tsec, int row, int col, int nIter)
{
        float gflops=(1.0e-9 * (( 1.0 * row*col )*nIter/Tsec));
	return gflops;
}

/*sequential function for mat vect multiplication*/
void CPU_MatVect()
{
	cpu_ResVect = (double *)malloc(matRowSize*sizeof(double));
	if(cpu_ResVect==NULL)
                mem_error("cpu_ResVect","vectmatmul",size,"double");

	int i,j;
	for(i=0;i<matRowSize;i++)
	{cpu_ResVect[i]=0;
	for(j=0;j<matColSize;j++)
	cpu_ResVect[i]+=host_Mat[i*vlength+j]*host_Vect[j];
	}
}

/*Check for safe return of all calls to the device */
void CUDA_SAFE_CALL(hipError_t call)
{
        hipError_t ret = call;
        //printf("RETURN FROM THE CUDA CALL:%d\t:",ret);                                        
        switch(ret)
        {
                case hipSuccess:
                //              printf("Success\n");                    
                                break;
        /*      case hipErrorInvalidValue:                             
                                {
                                printf("ERROR: InvalidValue:%i.\n",__LINE__);
                                exit(-1);
                                break;  
                                }                       
                case hipErrorInvalidDevicePointer:                     
                                {
                                printf("ERROR:Invalid Device pointeri:%i.\n",__LINE__);
                                exit(-1);
                                break;
                                }                       
                case hipErrorInvalidMemcpyDirection:                   
                                {
                                printf("ERROR:Invalid memcpy direction:%i.\n",__LINE__);        
                                exit(-1);
                                break;
                                }                       */
                default:
                        {
                                printf(" ERROR at line :%i.%d' ' %s\n",__LINE__,ret,hipGetErrorString(ret));
                                exit(-1);
                                break;
                        }
        }
}
 

/*free memory*/
void dfree(double * arr[],int len)
{
        for(int i=0;i<len;i++)
                CUDA_SAFE_CALL(hipFree(arr[i]));
        printf("mem freed\n");
}

/* function to calculate relative error*/
void relError(double* dRes,double* hRes,int size)
{
        double relativeError=0.0,errorNorm=0.0;
        int flag=0;
        int i;
        

        for( i = 0; i < size; ++i) {
                if (fabs(hRes[i]) > fabs(dRes[i]))
                        relativeError = fabs((hRes[i] - dRes[i]) / hRes[i]);
                else
                        relativeError = fabs((dRes[i] - hRes[i]) / dRes[i]);

                if (relativeError > EPS && relativeError != 0.0e+00 )
                {
                        if(errorNorm < relativeError)
                        {
                                errorNorm = relativeError;
                                flag=1;
                        }
                }

        }
        if( flag == 1)
        {
                printf(" \n Results verfication : Failed");
                printf(" \n Considered machine precision : %e", EPS);
                printf(" \n Relative Error                  : %e\n", errorNorm);

        }
        else
                printf("\n Results verfication : Success\n");

}


/*prints the result in screen*/
void print_on_screen(char * program_name,float tsec,double gflops,int row, int col, int flag)//flag=1 if gflops has been calculated else flag =0
{
        printf("\n---------------%s----------------\n",program_name);
        printf("\tSIZE\t TIME_SEC\t Gflops\n");
        if(flag==1)
        printf("\t%d,%d\t%f\t%lf\t",row, col,tsec,gflops);
        else
        printf("\t%d,%d\t%lf\t%lf\t",row, col,"---","---");

}

/*funtion to check blocks per grid and threads per block*/
void check_block_grid_dim(hipDeviceProp_t devProp,dim3 blockDim,dim3 gridDim)
{

        if( blockDim.x >= devProp.maxThreadsDim[0] || blockDim.y >= devProp.maxThreadsDim[1] || blockDim.z >= devProp.maxThreadsDim[2] )
        {
                printf("\nBlock Dimensions exceed the maximum limits:%d * %d * %d \n",devProp.maxThreadsDim[0],devProp.maxThreadsDim[1],devProp.maxThreadsDim[2]);
               exit(-1);
        }

        if( gridDim.x >= devProp.maxGridSize[0] || gridDim.y >= devProp.maxGridSize[1] || gridDim.z >= devProp.maxGridSize[2] )
        {
                printf("\nGrid Dimensions exceed the maximum limits:%d * %d * %d \n",devProp.maxGridSize[0],devProp.maxGridSize[1],devProp.maxGridSize[2]);
               exit(-1);
        }
}


/*Get the number of GPU devices present on the host */
int get_DeviceCount()
{
        int count;
        hipGetDeviceCount(&count);
        return count;
}


/*Fill in the vector with double precision values */
void fill_dp_vector(double* vec,int size)
{
        int ind;
        for(ind=0;ind<size;ind++)
                vec[ind]=drand48();
}


/////////////////////////////////////////////////////////////////////////////////////////
//
// MatVect : this kernel will perform actual MatrixVector Multiplication 
//
/////////////////////////////////////////////////////////////////////////////////////////
__global__ void MatVectMultiplication(double *device_Mat, double *device_Vect,int matRowSize, int vlength,double *device_ResVect)
  {
        //int tidx = blockIdx.x*blockDim.x + threadIdx.x;
        //int tidy = blockIdx.y*blockDim.y + threadIdx.y;
        //int tindex=tidx+gridDim.x*BLOCKSIZE*tidy;
        int tindex = blockIdx.x*blockDim.x + threadIdx.x;


        //if(tindex<matRowSize)
	    //{
                int i;int m=tindex*vlength;
	        device_ResVect[tindex]=0;
	        for(i=0;i<vlength;i++)
	             device_ResVect[tindex]+=device_Mat[m+i]*device_Vect[i];
	    //}

     __syncthreads();

  }//end of MatVect device function



/*function to launch kernel*/
void launch_Kernel_MatVectMul()
{
/*          threads_per_block, blocks_per_grid  */


    

}


/*main function*/
int main(int argc, char* argv[])
{
	// Vector length , Matrix Row and Col sizes..............
       	vlength = matColSize = atoi(argv[2]);
       	matRowSize =atoi(argv[1]);
    
     	//  printf("this programs does computation of square matrix only\n");
	float elapsedTime,Tsec;
	hipEvent_t start,stop;

	device_Count=get_DeviceCount();
        printf("\n\nNUmber of Devices : %d\n\n", device_Count);

        // Device Selection, Device 1: Tesla C1060
        hipSetDevice(0);
      
        int device;
        // Current Device Detection
        hipGetDevice(&device);
        hipGetDeviceProperties(&deviceProp,device);
        printf("Using device %d: %s \n", device, deviceProp.name);


	
  	/*allocating the memory for each matrix */
	host_Mat =new double[matRowSize*matColSize];
	host_Vect = new double[vlength];
	host_ResVect = new double[matRowSize];

	
	// ---------------checking host memory  for error..............................
	 if(host_Mat==NULL)
                mem_error("host_Mat","vectmatmul",matRowSize*matColSize,"double");

         if(host_Vect==NULL)
                mem_error("host_Vect","vectmatmul",vlength,"double");

         if(host_ResVect==NULL)
                mem_error("host_ResVect","vectmatmul",matRowSize,"double");

	//--------------Initializing the input arrays..............
    srand48(time(nullptr));
	fill_dp_vector(host_Mat,matRowSize*matColSize);
	fill_dp_vector(host_Vect,vlength);

    // //print
    //     printf("host_mat:");
    //     for(int k=0; k<matRowSize*matColSize; k++){
    //         printf("%f ",host_Mat[k]);
    //     }
    //     printf("\n");

    //     printf("host_Vect:");
    //     for(int k=0; k<vlength; k++){
    //         printf("%f ",host_Vect[k]);
    //     }
    //     printf("\n");

 	/* allocate memory for GPU events 
        start = (hipEvent_t) malloc (sizeof(hipEvent_t));
        stop = (hipEvent_t) malloc (sizeof(hipEvent_t));
        if(start==NULL)
                mem_error("start","vectvectmul",1,"hipEvent_t");
        if(stop==NULL)
                mem_error("stop","vectvectmul",1,"hipEvent_t");*/
  	
	//event creation...
        CUDA_SAFE_CALL(hipEventCreate (&start));
        CUDA_SAFE_CALL(hipEventCreate (&stop));

  	//allocating memory on GPU
	CUDA_SAFE_CALL(hipMalloc( (void**)&device_Mat, matRowSize*matColSize* sizeof(double)));
	CUDA_SAFE_CALL(hipMalloc( (void**)&device_Vect, vlength* sizeof(double)));
	CUDA_SAFE_CALL(hipMalloc( (void**)&device_ResVect, matRowSize* sizeof(double)));

 	//moving data from CPU to GPU
	CUDA_SAFE_CALL(hipMemcpy((void*)device_Mat, (void*)host_Mat, matRowSize*matColSize*sizeof(double) ,hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy((void*)device_Vect, (void*)host_Vect,vlength*sizeof(double),hipMemcpyHostToDevice));

	// Launching kernell..........	
	CUDA_SAFE_CALL(hipEventRecord (start, 0));
	
	int BlocksPerGrid=(matRowSize+BLOCKSIZE-1)/BLOCKSIZE;
        dim3 dimBlock(BLOCKSIZE);
        dim3 dimGrid(BlocksPerGrid);
        //check_block_grid_dim(deviceProp,dimBlock,dimGrid);

        int nIter = 300;
        for(int i=0; i<nIter; i++)
                MatVectMultiplication<<<(matRowSize+BLOCKSIZE-1)/BLOCKSIZE,BLOCKSIZE>>>(device_Mat,device_Vect,matRowSize,vlength,device_ResVect);
	
	CUDA_SAFE_CALL(hipEventRecord (stop, 0));
	CUDA_SAFE_CALL(hipEventSynchronize (stop));
	CUDA_SAFE_CALL(hipEventElapsedTime ( &elapsedTime, start, stop));

	Tsec= 1.0e-3*elapsedTime;
 	
	// calling funtion for measuring Gflops
        //calculate_gflops(Tsec, matRowSize, matColSize, nIter);
	
	//printing the result on screen
    	print_on_screen("MAT VECT MULTIPLICATION",Tsec,calculate_gflops(Tsec, matRowSize, matColSize, nIter),matRowSize, matColSize,1); 

 
	//retriving result from device
  	CUDA_SAFE_CALL(hipMemcpy((void*)host_ResVect, (void*)device_ResVect,matRowSize*sizeof(double),hipMemcpyDeviceToHost));

	// CPU calculation..and checking error deviation....
	CPU_MatVect();
    // //print
    //     printf("\n");
    //     printf("host:");
    //     for(int k=0; k<size; k++){
    //         printf("%f ",host_ResVect[k]);
    //     }
    //     printf("\n");

    //     printf("cpu:");
    //     for(int k=0; k<size; k++){
    //         printf("%f ",cpu_ResVect[k]);
    //     }
    //     printf("\n");
  	relError(cpu_ResVect,host_ResVect,size);
   	printf("\n ----------------------------------------------------------------------\n");

	/*free the memory from GPU */
	double *array[3];
        array[0]=device_Mat;
        array[1]=device_Vect;
        array[2]=device_ResVect;
        dfree(array,3);

	//free host memory----------
        free(host_Mat);
        free(host_Vect);
        free(host_ResVect);
        free(cpu_ResVect);

	return 0;
}// end of main